
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

int main() {
  int d;
  hipGetDevice(&d);

  int pma = 0;
  hipDeviceGetAttribute(&pma, hipDeviceAttributePageableMemoryAccess, d);
  printf("Full Unified Memory Support: %s\n", pma == 1? "YES" : "NO");
  
  int cma = 0;
  hipDeviceGetAttribute(&cma, hipDeviceAttributeConcurrentManagedAccess, d);
  printf("CUDA Managed Memory with full support: %s\n", cma == 1? "YES" : "NO");

  return 0;
}